#include "hip/hip_runtime.h"
#include <starpu.h>
#include <starpu_cuda.h>

static __global__ void myf(int *dMatA)
{
  int tidy = threadIdx.y;

  dMatA[ tidy ] = dMatA[ tidy ]  * 5;
}

extern "C" void my_codelet_gpu(void *descr[], void *_args)
{
  unsigned nx = STARPU_VECTOR_GET_NX(descr[0]);
  int *sub = (int *)STARPU_VECTOR_GET_PTR(descr[0]);

  dim3 dimGrid(1,1);
  dim3 dimBlock(nx,nx);

  myf<<<dimGrid, dimBlock, 0, starpu_cuda_get_local_stream()>>>(sub);
 
  hipStreamSynchronize(starpu_cuda_get_local_stream());
}
